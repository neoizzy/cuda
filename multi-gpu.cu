#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main(void) {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount < 2) {
        printf("This program requires at least 2 GPUs.\n");
        return 1;
    }

    int N = 1 << 20;       // Total elements (1M)
    int halfN = N / 2;     // Elements per GPU
    size_t totalSize = N * sizeof(float);
    size_t halfSize = halfN * sizeof(float);

    // Allocate host memory
    float *h_A = (float*)malloc(totalSize);
    float *h_B = (float*)malloc(totalSize);
    float *h_C = (float*)malloc(totalSize);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Device pointers for GPU 0
    float *d_A0, *d_B0, *d_C0;
    hipSetDevice(0);
    hipMalloc((void**)&d_A0, halfSize);
    hipMalloc((void**)&d_B0, halfSize);
    hipMalloc((void**)&d_C0, halfSize);

    // Device pointers for GPU 1
    float *d_A1, *d_B1, *d_C1;
    hipSetDevice(1);
    hipMalloc((void**)&d_A1, halfSize);
    hipMalloc((void**)&d_B1, halfSize);
    hipMalloc((void**)&d_C1, halfSize);

    // Copy data to each GPU
    hipSetDevice(0);
    hipMemcpy(d_A0, h_A, halfSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B0, h_B, halfSize, hipMemcpyHostToDevice);

    hipSetDevice(1);
    hipMemcpy(d_A1, h_A + halfN, halfSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B1, h_B + halfN, halfSize, hipMemcpyHostToDevice);

    // Create CUDA events for timing on each GPU
    hipEvent_t start0, stop0;
    hipEventCreate(&start0);
    hipEventCreate(&stop0);

    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);

    int threadsPerBlock = 256;
    int blocksPerGrid = (halfN + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel on GPU 0
    hipSetDevice(0);
    hipEventRecord(start0, 0);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A0, d_B0, d_C0, halfN);
    hipEventRecord(stop0, 0);

    // Launch kernel on GPU 1
    hipSetDevice(1);
    hipEventRecord(start1, 0);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A1, d_B1, d_C1, halfN);
    hipEventRecord(stop1, 0);

    // Synchronize both GPUs
    hipSetDevice(0);
    hipEventSynchronize(stop0);
    hipSetDevice(1);
    hipEventSynchronize(stop1);

    // Measure elapsed time on each GPU
    float elapsedTime0, elapsedTime1;
    hipSetDevice(0);
    hipEventElapsedTime(&elapsedTime0, start0, stop0);
    hipSetDevice(1);
    hipEventElapsedTime(&elapsedTime1, start1, stop1);

    printf("GPU 0 vector addition took: %f ms\n", elapsedTime0);
    printf("GPU 1 vector addition took: %f ms\n", elapsedTime1);

    // The overall runtime is approximately the maximum of the two times
    float overallTime = (elapsedTime0 > elapsedTime1) ? elapsedTime0 : elapsedTime1;
    printf("Overall runtime (max of both GPUs): %f ms\n", overallTime);

    // Copy results back to host
    hipSetDevice(0);
    hipMemcpy(h_C, d_C0, halfSize, hipMemcpyDeviceToHost);
    hipSetDevice(1);
    hipMemcpy(h_C + halfN, d_C1, halfSize, hipMemcpyDeviceToHost);

    // (Optional) Verify the results
    for (int i = 0; i < N; i++) {
        if (fabs(h_C[i] - 3.0f) > 1e-5) {
            printf("Error at index %d: %f != 3.0\n", i, h_C[i]);
            break;
        }
    }

    // Print h_C array (first 10 elements)
    printf("h_C array (first 10 elements):\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_C[i]);
    }
    printf("\n");

    // Cleanup GPU 0
    hipSetDevice(0);
    hipFree(d_A0);
    hipFree(d_B0);
    hipFree(d_C0);
    hipEventDestroy(start0);
    hipEventDestroy(stop0);

    // Cleanup GPU 1
    hipSetDevice(1);
    hipFree(d_A1);
    hipFree(d_B1);
    hipFree(d_C1);
    hipEventDestroy(start1);
    hipEventDestroy(stop1);

    // Cleanup host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
