// management of multiple gpus (with and withoug groupstart/groupend)
 
#include <cstdio>
#include <cstdlib>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define NUM_DEVICES 2
#define COUNT 1024

// Error-checking macros.
#define CUDACHECK(cmd) do {                                 \
    hipError_t e = cmd;                                  \
    if( e != hipSuccess ) {                              \
        printf("Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                               \
    }                                                     \
} while(0)

#define NCCLCHECK(cmd) do {                                 \
    ncclResult_t r = cmd;                                  \
    if (r != ncclSuccess) {                                \
        printf("NCCL error %s:%d '%s'\n", __FILE__, __LINE__, ncclGetErrorString(r)); \
        exit(EXIT_FAILURE);                               \
    }                                                     \
} while(0)

int main() {
    int nDevices = NUM_DEVICES;
    int devices[NUM_DEVICES] = {0, 1};
    ncclComm_t comms[NUM_DEVICES];
    hipStream_t streams[NUM_DEVICES];
    float* d_buffers[NUM_DEVICES];

    // --------------------------
    // NCCL Communicator Setup
    // --------------------------
    // Here we use ncclCommInitAll to initialize communicators for all local GPUs.
    NCCLCHECK(ncclCommInitAll(comms, nDevices, devices));

    // --------------------------
    // Allocate device memory and create streams.
    // --------------------------
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        CUDACHECK(hipStreamCreate(&streams[i]));
        CUDACHECK(hipMalloc(&d_buffers[i], COUNT * sizeof(float)));
        // Initialize the device memory to zeros.
        CUDACHECK(hipMemset(d_buffers[i], 0, COUNT * sizeof(float)));
    }

    // --------------------------
    // Create CUDA events for timing.
    // --------------------------
    hipEvent_t start, stop;
    CUDACHECK(hipEventCreate(&start));
    CUDACHECK(hipEventCreate(&stop));

    float timeWithoutGroup = 0.0f;
    float timeWithGroup = 0.0f;

    // ****************** Without Grouping ****************** //
    // Each NCCL call is issued one-by-one.
    CUDACHECK(hipEventRecord(start, 0));
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        NCCLCHECK(ncclAllReduce(
            (const void*)d_buffers[i],
            (void*)d_buffers[i],
            COUNT,
            ncclFloat,
            ncclSum,
            comms[i],
            streams[i]
        ));
    }
    // Ensure that all operations have completed.
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        CUDACHECK(hipStreamSynchronize(streams[i]));
    }
    CUDACHECK(hipEventRecord(stop, 0));
    CUDACHECK(hipEventSynchronize(stop));
    CUDACHECK(hipEventElapsedTime(&timeWithoutGroup, start, stop));
    printf("Time without group: %f ms\n", timeWithoutGroup);

    // ****************** With Grouping ****************** //
    // The NCCL group call allows concurrent enqueuing across devices.
    CUDACHECK(hipEventRecord(start, 0));
    ncclGroupStart();
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        NCCLCHECK(ncclAllReduce(
            (const void*)d_buffers[i],
            (void*)d_buffers[i],
            COUNT,
            ncclFloat,
            ncclSum,
            comms[i],
            streams[i]
        ));
    }
    ncclGroupEnd();
    // Synchronize all streams.
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        CUDACHECK(hipStreamSynchronize(streams[i]));
    }
    CUDACHECK(hipEventRecord(stop, 0));
    CUDACHECK(hipEventSynchronize(stop));
    CUDACHECK(hipEventElapsedTime(&timeWithGroup, start, stop));
    printf("Time with group: %f ms\n", timeWithGroup);

    // --------------------------
    // Cleanup: Free device memory, destroy streams and communicators.
    // --------------------------
    for (int i = 0; i < nDevices; i++) {
        CUDACHECK(hipSetDevice(devices[i]));
        CUDACHECK(hipFree(d_buffers[i]));
        CUDACHECK(hipStreamDestroy(streams[i]));
        NCCLCHECK(ncclCommDestroy(comms[i]));
    }
    CUDACHECK(hipEventDestroy(start));
    CUDACHECK(hipEventDestroy(stop));

    return 0;
}
