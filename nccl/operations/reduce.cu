#include <cstdio>
#include <cstdlib>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define N 4  // Number of elements in each buffer

// Macro to check for CUDA errors.
#define CHECK_CUDA(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

// Macro to check for NCCL errors.
#define CHECK_NCCL(call)                                      \
    do {                                                      \
        ncclResult_t res = call;                              \
        if (res != ncclSuccess) {                             \
            fprintf(stderr, "NCCL error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, ncclGetErrorString(res)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

int main() {
    int numDevices = 0;
    CHECK_CUDA(hipGetDeviceCount(&numDevices));

    if (numDevices < 2) {
        printf("This example requires at least two GPUs.\n");
        return 0;
    }

    // For this example, we use 2 GPUs.
    const int numRanks = 2;
    int devices[numRanks] = {0, 1};

    // Define the root rank for the reduction.
    int root = 0;

    ncclComm_t comms[numRanks];
    float* sendbuffs[numRanks];
    float* recvbuffs[numRanks];
    hipStream_t streams[numRanks];

    // Allocate memory and create streams on each GPU.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipMalloc(&sendbuffs[i], N * sizeof(float)));
        // Allocate recvbuff even if it is only used on the root.
        CHECK_CUDA(hipMalloc(&recvbuffs[i], N * sizeof(float)));
        CHECK_CUDA(hipStreamCreate(&streams[i]));

        // Initialize the send buffer with sample data:
        // GPU 0 gets [1, 1, 1, 1] and GPU 1 gets [2, 2, 2, 2].
        float data[N];
        for (int j = 0; j < N; ++j)
            data[j] = (i == root) ? 1.0f : 2.0f;
        CHECK_CUDA(hipMemcpy(sendbuffs[i], data, N * sizeof(float), hipMemcpyHostToDevice));
    }

    // Initialize NCCL communicators across the GPUs.
    CHECK_NCCL(ncclCommInitAll(comms, numRanks, devices));

    // Perform the Reduce operation.
    // Only the root rank (rank 0) will receive the reduced result.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_NCCL(ncclReduce(
            /* sendbuff */ (const void*)sendbuffs[i],
            /* recvbuff */ (void*)recvbuffs[i],
            /* count */ N,
            /* datatype */ ncclFloat,
            /* op */ ncclSum,
            /* root */ root,
            /* communicator */ comms[i],
            /* stream */ streams[i]
        ));
    }

    // Synchronize streams to ensure the reduce operation is complete.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

    // Only the root rank has the valid reduced result.
    if (root < numRanks) {
        float hostResult[N];
        CHECK_CUDA(hipMemcpy(hostResult, recvbuffs[root], N * sizeof(float), hipMemcpyDeviceToHost));
        printf("Reduced result on root (device %d): ", devices[root]);
        for (int j = 0; j < N; ++j)
            printf("%f ", hostResult[j]);
        printf("\n");
    }

    // Cleanup: free allocated GPU memory, destroy streams and NCCL communicators.
    for (int i = 0; i < numRanks; ++i) {
        ncclCommDestroy(comms[i]);
        CHECK_CUDA(hipFree(sendbuffs[i]));
        CHECK_CUDA(hipFree(recvbuffs[i]));
        CHECK_CUDA(hipStreamDestroy(streams[i]));
    }

    return 0;
}
