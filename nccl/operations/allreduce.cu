#include <cstdio>
#include <cstdlib>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define N 4 // Number of elements per GPU
#define CHECK_CUDA(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

#define CHECK_NCCL(call)                                      \
    do {                                                      \
        ncclResult_t res = call;                              \
        if (res != ncclSuccess) {                             \
            fprintf(stderr, "NCCL error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, ncclGetErrorString(res)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

int main() {
    int numDevices = 0;
    CHECK_CUDA(hipGetDeviceCount(&numDevices));

    if (numDevices < 2) {
        printf("This example requires at least two GPUs.\n");
        return 0;
    }

    const int numRanks = 2; // Using 2 GPUs for this example.
    int devices[numRanks] = {0, 1}; // Select which GPUs to use.

    ncclComm_t comms[numRanks];
    float* sendbuffs[numRanks];
    float* recvbuffs[numRanks];
    hipStream_t streams[numRanks];

    // Allocate memory and create streams on each GPU.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipMalloc(&sendbuffs[i], N * sizeof(float)));
        CHECK_CUDA(hipMalloc(&recvbuffs[i], N * sizeof(float)));
        CHECK_CUDA(hipStreamCreate(&streams[i]));

        // Initialize send buffer with sample data.
        float data[N];
        for (int j = 0; j < N; ++j)
            data[j] = (float)(i + 1); // For example: GPU 0 gets [1,1,1,1], GPU 1 gets [2,2,2,2]
        CHECK_CUDA(hipMemcpy(sendbuffs[i], data, N * sizeof(float), hipMemcpyHostToDevice));
    }

    // Initialize NCCL communicators for all GPUs.
    CHECK_NCCL(ncclCommInitAll(comms, numRanks, devices));

    // Perform the AllReduce operation on each GPU.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_NCCL(ncclAllReduce((const void*)sendbuffs[i],
                                 (void*)recvbuffs[i],
                                 N,
                                 ncclFloat,
                                 ncclSum,
                                 comms[i],
                                 streams[i]));
    }

    // Synchronize streams to ensure completion.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }

    // Retrieve and print the results from each GPU.
    for (int i = 0; i < numRanks; ++i) {
        float result[N];
        CHECK_CUDA(hipMemcpy(result, recvbuffs[i], N * sizeof(float), hipMemcpyDeviceToHost));
        printf("Device %d result: ", devices[i]);
        for (int j = 0; j < N; ++j)
            printf("%f ", result[j]);
        printf("\n");
    }

    // Cleanup: Destroy communicators, free memory, and destroy streams.
    for (int i = 0; i < numRanks; ++i) {
        ncclCommDestroy(comms[i]);
        hipFree(sendbuffs[i]);
        hipFree(recvbuffs[i]);
        hipStreamDestroy(streams[i]);
    }

    return 0;
}
