#include <cstdio>
#include <cstdlib>
#include <nccl.h>
#include <hip/hip_runtime.h>

#define N 4  // Number of elements in the buffer

// Macro to check for CUDA errors.
#define CHECK_CUDA(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

// Macro to check for NCCL errors.
#define CHECK_NCCL(call)                                      \
    do {                                                      \
        ncclResult_t res = call;                              \
        if (res != ncclSuccess) {                             \
            fprintf(stderr, "NCCL error at %s:%d: %s\n",      \
                    __FILE__, __LINE__, ncclGetErrorString(res)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

int main() {
    int numDevices = 0;
    CHECK_CUDA(hipGetDeviceCount(&numDevices));
    
    if (numDevices < 2) {
        printf("This example requires at least two GPUs.\n");
        return 0;
    }
    
    // For simplicity, we'll use 2 GPUs.
    const int numRanks = 2;
    int devices[numRanks] = {0, 1};
    
    // Set the root rank for the broadcast.
    int root = 0;
    
    ncclComm_t comms[numRanks];
    float* sendbuffs[numRanks];
    float* recvbuffs[numRanks];
    hipStream_t streams[numRanks];
    
    // Allocate memory and create streams on each GPU.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipMalloc(&sendbuffs[i], N * sizeof(float)));
        CHECK_CUDA(hipMalloc(&recvbuffs[i], N * sizeof(float)));
        CHECK_CUDA(hipStreamCreate(&streams[i]));
        
        // Only the root rank initializes its send buffer with data.
        if (i == root) {
            float data[N];
            for (int j = 0; j < N; ++j) {
                data[j] = (float)(j + 1);  // For example: [1, 2, 3, 4]
            }
            CHECK_CUDA(hipMemcpy(sendbuffs[i], data, N * sizeof(float), hipMemcpyHostToDevice));
        }
    }
    
    // Initialize NCCL communicators.
    CHECK_NCCL(ncclCommInitAll(comms, numRanks, devices));
    
    // Perform the broadcast operation.
    // The root rank's sendbuff is used, and all GPUs receive the broadcast data in their recvbuff.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_NCCL(ncclBroadcast(
            /* sendbuff */ (const void*)sendbuffs[root],
            /* recvbuff */ (void*)recvbuffs[i],
            /* count */ N,
            /* datatype */ ncclFloat,
            /* root */ root,
            /* communicator */ comms[i],
            /* stream */ streams[i]
        ));
    }
    
    // Wait for all broadcast operations to complete.
    for (int i = 0; i < numRanks; ++i) {
        CHECK_CUDA(hipSetDevice(devices[i]));
        CHECK_CUDA(hipStreamSynchronize(streams[i]));
    }
    
    // Copy the data back to the host and print the results.
    for (int i = 0; i < numRanks; ++i) {
        float hostData[N];
        CHECK_CUDA(hipMemcpy(hostData, recvbuffs[i], N * sizeof(float), hipMemcpyDeviceToHost));
        printf("Device %d received data: ", devices[i]);
        for (int j = 0; j < N; ++j)
            printf("%f ", hostData[j]);
        printf("\n");
    }
    
    // Cleanup: free GPU memory, destroy streams and NCCL communicators.
    for (int i = 0; i < numRanks; ++i) {
        ncclCommDestroy(comms[i]);
        CHECK_CUDA(hipFree(sendbuffs[i]));
        CHECK_CUDA(hipFree(recvbuffs[i]));
        CHECK_CUDA(hipStreamDestroy(streams[i]));
    }
    
    return 0;
}
